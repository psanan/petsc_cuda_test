#include "hip/hip_runtime.h"
/* Test for a simple CUDA kernel which simply sets an array to a constant value */
#include <petscsystypes.h>

__global__ void set_constant_value(PetscScalar *device_array, PetscInt n, PetscScalar value)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) device_array[i] = value;
}

extern "C"
PetscErrorCode set_device_array_constant(PetscScalar *device_array, PetscInt n, PetscScalar value)
{
  set_constant_value<<<(n+127)/128,128>>>(device_array, n, value);
  return 0;
}
